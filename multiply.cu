
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

// CUDA RumTime API

void MatrixMultiplyOnHost(float* M, float* N, float* P, int width)
{
	for(int i=0; i<width; ++i)
	{
		for (int j=0; j<width; ++j)
		{
			float sum = 0;
			for(int k=0; k<width; ++k)
			{
				float a = M[i*width+k];
				float b = N[k*width+j];
				sum += a*b;
			}
			P[i*width+j] = sum;
		}
	}

}

__global__ void MatirxMultiplyKernel(const float* devM, const float* devN, float* devP, const int width)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//Initialize accumulator to 0
	float pValue = 0;

	//Multiply and add
	for(int k=0; k<width; k++)
	{
		float m = devM[ty*width+k];
		float n = devN[k*width+tx];
		pValue += m*n;
	}

	//Write value to device memory - each thread has unique index to write to
	devP[ty*width+tx] = pValue;
}

void MatrixMultiplyOnDevice(const float* hostM, const float* hostN, float* hostP, const int width)
{
	int sizeInBytes = width*width*sizeof(float);
	float *devM, *devN, *devP;

	//Allocate M and N on devide
	hipMalloc((void**)&devM, sizeInBytes);
	hipMalloc((void**)&devN, sizeInBytes);

	//Allocate P
	hipMalloc((void**)&devP, sizeInBytes);

	//Copy M and N from host to device
	hipMemcpy(devM, hostM, sizeInBytes, hipMemcpyHostToDevice);
	hipMemcpy(devN, hostN, sizeInBytes, hipMemcpyHostToDevice);

	//Setup thread/block execution configuration
	dim3 threads(width,width); //Each block has (width,width) threads
	dim3 blocks(1,1); //Launch 1 block


	//Launch the kernel
	clock_t begin = clock();
	MatirxMultiplyKernel<<<blocks,threads>>>(devM,devN,devP,width);
	clock_t end = clock();
	float elapsed_secs = float(end - begin) / CLOCKS_PER_SEC;
	printf("Matrix Multiply on Device: %fs\n",elapsed_secs);
	

	//Copy P matrix from device to host
	hipMemcpy(hostP, devP, sizeInBytes, hipMemcpyDeviceToHost);

	//Free allocated memory
	hipFree(devM); hipFree(devN); hipFree(devP);
}

void PrintMatrix(float* M, int width)
{
	for(int i=0; i<width; i++)
	{
		for(int j=0; j<width; j++)
		{
			printf("%f ",M[i*width+j]);
		}
		printf("\n");
	}
}

int main()
{
	int width = 32;
	int size = width*width;

	float* M = new float[size];
	float* N = new float[size];
	float* P = new float[size];
	float* Q = new float[size];
	srand (time(NULL));
	for(int i=0; i<size; i++)
	{
		M[i] = rand() / (RAND_MAX + 1.);
		N[i] = rand() / (RAND_MAX + 1.);
	}

	//multiply on host
	clock_t begin = clock();
	MatrixMultiplyOnHost(M,N,P,width);
	clock_t end = clock();
	float elapsed_secs = float(end - begin) / CLOCKS_PER_SEC;
	printf("Matrix Multiply on Host: %fs\n",elapsed_secs);
	//std::cout << "Matrix Multiply on Host: " << elapsed_secs << std::endl;
	
	//multiply on device
	//1. Copy M,N matrices to device
	//2. M*N on device
	//3. Copy P matrix to host and output
	MatrixMultiplyOnDevice(M,N,Q,width);

	float avg_err = 0;
	for(int i=0; i<size; i++)
		avg_err += fabs(P[i]-Q[i]);
	avg_err /= size;
	printf("Average error is: %f\n",avg_err);
	//PrintMatrix(M,width);
	//PrintMatrix(N,width);
	//PrintMatrix(P,width);
	//PrintMatrix(Q,width);

	return 0;
}